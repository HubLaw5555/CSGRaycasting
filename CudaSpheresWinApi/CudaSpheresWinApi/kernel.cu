#include "hip/hip_runtime.h"
﻿#include "render.cuh"

LRESULT CALLBACK WndProc(HWND hwnd, UINT msg, WPARAM wParam, LPARAM lParam);
void MouseMove(HWND hwnd, WPARAM wp, LPARAM lp);
void KeyPress(int keyCode);
void initBackBuffer(HWND hwnd);
void initialise_csg();
void initialize_memory();
void copy_scene_to_gpu(int n);
void move_camera(HWND hwnd);

HDC hBackDC = NULL;
HBITMAP hBackBitmap = NULL;


hipEvent_t start_render, stop_render;
hipEvent_t start_gpu_cpu, stop_gpu_cpu;
hipEvent_t start_cpu_gpu, stop_cpu_gpu;

float time_render = .0f, time_cpu_gpu = .0f, time_gpu_cpu = .0f;
int gpu_cpu_cnt = 1;


unsigned int* cuda_buffer;
unsigned int* cpu_buffer;

camera gpu_cam;
light gpu_lights, cpu_lights;
csg_scene cpu_scene, gpu_scene;

float curr_x = .0f, curr_y = .0f;
float prev_x = 0, prev_y = 0;

float3 shift = make_float3(0, 0, 0);

int reshaped_width = 0;
int reshaped_height = 0;
bool current_reshape = false;


// now here, after that in globals
const int sphere_cnt = 2;
const int lvls = 2;

void initialize_memory()
{
	cpu_buffer = new unsigned int[WIDTH * HEIGHT];
	memset(cpu_buffer, 0, sizeof(unsigned int) * WIDTH * HEIGHT);

	checkCudaErrors(hipMalloc((void**)&cuda_buffer, WIDTH * HEIGHT * sizeof(unsigned int)));

	initialise_csg();
	cpu_scene.calculate_bounding_boxes();
	float x1 = cpu_scene.bounding.pos.x[0];
	float y1 = cpu_scene.bounding.pos.y[0];
	float z1 = cpu_scene.bounding.pos.z[0];
	float x2 = cpu_scene.bounding.pos.x[1];
	float y2 = cpu_scene.bounding.pos.y[1];
	float z2 = cpu_scene.bounding.pos.z[1];
	float x3 = cpu_scene.bounding.pos.x[2];
	float y3 = cpu_scene.bounding.pos.y[2];
	float z3 = cpu_scene.bounding.pos.z[2];

	float r1 = cpu_scene.bounding.radius[0];
	float r2 = cpu_scene.bounding.radius[1];
	float r3 = cpu_scene.bounding.radius[2];
	copy_scene_to_gpu(2);

	gpu_cam = camera(make_float3(0, 0.7f, 10.7f), make_float3(0, 0, -1.0f) /*make_float3(0,0,0), make_float3(0,0,1)*/,
		make_float3(0.0f, 1.0f, 0.0f), 30.0f, RATIO);
}

void draw(HWND hwnd)
{

	dim3 blocksCount = dim3(WIDTH, HEIGHT);
	//dim3 threadsCount = dim3(, framesY);
	float time;

	int bigPow = pow(2, lvls) - 1;
	int shared_size = 2 * bigPow * sizeof(float) + bigPow * sizeof(int) + sphere_cnt * sizeof(float3);
	render <<<blocksCount, 32, shared_size >>> (cuda_buffer, gpu_cam, gpu_lights, gpu_scene, WIDTH, HEIGHT);
	hipDeviceSynchronize();
	hipEventRecord(start_gpu_cpu, 0);
	checkCudaErrors(hipMemcpy(cpu_buffer, cuda_buffer, WIDTH * HEIGHT * sizeof(unsigned int), hipMemcpyDeviceToHost));
	hipEventRecord(stop_gpu_cpu, 0);
	hipEventSynchronize(stop_render);
	hipEventElapsedTime(&time, start_gpu_cpu, stop_gpu_cpu);
	time_gpu_cpu += time;

	SetBitmapBits(hBackBitmap, HEIGHT * WIDTH * sizeof(unsigned int), (const void*)(cpu_buffer));
	BitBlt(GetDC(hwnd), 0, 0, WIDTH, HEIGHT, hBackDC, 0, 0, SRCCOPY);
}

int WINAPI wWinMain(HINSTANCE hInstace, HINSTANCE hPrevInstace, LPWSTR lpCmdLine, int nCmdShow)
{

	MSG msg = { 0 };
	WNDCLASS wnd = { 0 };

	hipEventCreate(&start_render);
	hipEventCreate(&stop_render);
	hipEventCreate(&start_cpu_gpu);
	hipEventCreate(&stop_cpu_gpu);
	hipEventCreate(&start_gpu_cpu);
	hipEventCreate(&stop_gpu_cpu);

	wnd.lpfnWndProc = WndProc;
	wnd.hInstance = hInstace;
	wnd.lpszClassName = "Window";

	if (!RegisterClass(&wnd)) {
		return 0;
	}

	checkCudaErrors(hipSetDevice(0));
	initialize_memory();


	HWND hwnd = CreateWindowEx(WS_EX_CLIENTEDGE, wnd.lpszClassName, "Window",
		WS_OVERLAPPEDWINDOW, CW_USEDEFAULT, CW_USEDEFAULT, WIDTH, HEIGHT, NULL, NULL, hInstace, NULL);

	if (!hwnd) {
		return 0;
	}

	ShowWindow(hwnd, nCmdShow);
	UpdateWindow(hwnd);
	fps_counter fps;
	fps.n = 0;
	fps.fpsSum = .0f;

	while (true)
	{
		hipEventRecord(start_render, 0);

		if (PeekMessage(&msg, NULL, 0, 0, PM_REMOVE)) {
			if (msg.message == WM_QUIT) {
				break;
			}

			TranslateMessage(&msg);
			DispatchMessage(&msg);
		}

		move_camera(hwnd);
		draw(hwnd);

		hipEventRecord(stop_render, 0);
		hipEventSynchronize(stop_render);
		hipEventElapsedTime(&time_render, start_render, stop_render);
		std::string windText = " Average fps: " + std::to_string(fps.avg_fps(time_render));

		windText += " ms     CPU -> GPU copy: " + std::to_string(time_cpu_gpu) +
			" ms     Avg GPU -> CPU copy: " + std::to_string(time_gpu_cpu / float(gpu_cpu_cnt)) + " ms";
		gpu_cpu_cnt++;

		SetWindowText(hwnd, windText.c_str());
	}

	hipEventDestroy(start_render);
	hipEventDestroy(stop_render);
	hipEventDestroy(start_cpu_gpu);
	hipEventDestroy(stop_cpu_gpu);
	hipEventDestroy(start_gpu_cpu);
	hipEventDestroy(stop_gpu_cpu);

	return msg.wParam;
}


LRESULT CALLBACK WndProc(HWND hwnd, UINT msg, WPARAM wParam, LPARAM lParam) {

	wchar_t buffer[256];
	switch (msg) {
	case WM_CREATE:
		initBackBuffer(hwnd);
		break;
	case WM_DESTROY:
		DeleteDC(hBackDC);
		DeleteObject(hBackBitmap);
		PostQuitMessage(0);
		break;
	case WM_KEYDOWN:
		swprintf_s(buffer, 256, L"WM_KEYDOWN: 0x%x\n", wParam);
		KeyPress(wParam);
		OutputDebugStringW(buffer);
		break;

	case WM_KEYUP:
		swprintf_s(buffer, 256, L"WM_KEYUP: 0x%x\n", wParam);
		OutputDebugStringW(buffer);
		break;
	case WM_MOUSEMOVE:
		MouseMove(hwnd, wParam, lParam);
		break;
	}
	return DefWindowProc(hwnd, msg, wParam, lParam);
}

void KeyPress(int keyCode)
{
	switch (keyCode)
	{
	case 32: // space
		shift = shift + KEY_BATCH * (gpu_cam.vertical / gpu_cam.viewport_height);
		break;
	case 16: // shift
		shift = shift - KEY_BATCH * (gpu_cam.vertical / gpu_cam.viewport_height);
		break;
	case 0x57: // W
		shift = shift + KEY_BATCH * normalize(gpu_cam.look_at - gpu_cam.origin);
		break;
	case 0x53: // S
		shift = shift - KEY_BATCH * normalize(gpu_cam.look_at - gpu_cam.origin);
		break;
	case 0x41: // A
		shift = shift - KEY_BATCH * (gpu_cam.horizontal / gpu_cam.viewport_width);
		break;
	case 0x44: // D
		shift = shift + KEY_BATCH * (gpu_cam.horizontal / gpu_cam.viewport_width);
		break;
	}
}


void MouseMove(HWND hwnd, WPARAM wp, LPARAM lp)
{
	float x = LOWORD(lp);
	float y = HIWORD(lp);

	// to [-1,1]
	curr_x = 2.0f * x / WIDTH - 1.0f;
	curr_y = 2.0f * (HEIGHT - y) / HEIGHT - 1.0f;
}


void move_camera(HWND hwnd)
{
	gpu_cam.move_origin(shift);
	gpu_cam.move_look(MOUSE_SPEED * (curr_x - prev_x), MOUSE_SPEED * (curr_y - prev_y));


	prev_x = curr_x;
	prev_y = curr_y;
	shift = make_float3(0, 0, 0);
}

void initBackBuffer(HWND hwnd) {
	HDC hWinDC = GetDC(hwnd);
	BITMAPINFO bmi = { 0 };
	bmi.bmiHeader.biSize = sizeof(BITMAPCOREHEADER);
	bmi.bmiHeader.biWidth = WIDTH;
	bmi.bmiHeader.biHeight = -HEIGHT;
	bmi.bmiHeader.biPlanes = 1;
	bmi.bmiHeader.biBitCount = 32;
	bmi.bmiHeader.biCompression = BI_RGB;

	hBackDC = CreateCompatibleDC(hWinDC);
	hBackBitmap = CreateCompatibleBitmap(hWinDC, WIDTH, HEIGHT);
	SetBitmapBits(hBackBitmap, HEIGHT * WIDTH * sizeof(unsigned int), (const void*)(cpu_buffer));

	SelectObject(hBackDC, hBackBitmap);
	ReleaseDC(hwnd, hWinDC);
}


void initialise_csg()
{
	int m = LIGHT_COUNT;

	int levels = 2;
	int n = pow(2, levels - 1);
	int nodes = pow(2, levels) - 1;
	//scene allocation
	cpu_scene = csg_scene(levels);

	checkCudaErrors(hipMalloc((void**)&gpu_scene.objects.color.r, n * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_scene.objects.color.g, n * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_scene.objects.color.b, n * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_scene.objects.primitives.pos.x, n * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_scene.objects.primitives.pos.y, n * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_scene.objects.primitives.pos.z, n * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_scene.objects.primitives.radius, n * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_scene.objects.ka, n * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_scene.objects.kd, n * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_scene.objects.ks, n * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_scene.objects.alpha, n * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_scene.bounding.pos.x, nodes * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_scene.bounding.pos.y, nodes * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_scene.bounding.pos.z, nodes * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_scene.bounding.radius, nodes * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_scene.csg,  nodes* sizeof(int)));
	/*checkCudaErrors(hipMalloc((void**)&gpu_scene.levels, sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&gpu_scene.count, sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&gpu_scene.nodesCount, sizeof(int)));*/


	// lights alocation
	cpu_lights.is.r = new float[m];
	cpu_lights.is.g = new float[m];
	cpu_lights.is.b = new float[m];
	cpu_lights.id.r = new float[m];
	cpu_lights.id.g = new float[m];
	cpu_lights.id.b = new float[m];
	cpu_lights.pos.x = new float[m];
	cpu_lights.pos.y = new float[m];
	cpu_lights.pos.z = new float[m];

	checkCudaErrors(hipMalloc((void**)&gpu_lights.is.r, m * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_lights.is.g, m * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_lights.is.b, m * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_lights.id.r, m * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_lights.id.g, m * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_lights.id.b, m * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_lights.pos.x, m * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_lights.pos.y, m * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&gpu_lights.pos.z, m * sizeof(float)));

	for (int i = 0; i < m; ++i)
	{
		cpu_lights.is.r[i] = random_float(0.0f, 1.0f);
		cpu_lights.is.g[i] = random_float(0.0f, 1.0f);
		cpu_lights.is.b[i] = random_float(0.0f, 1.0f);
		cpu_lights.id.r[i] = random_float(0.0f, 1.0f);
		cpu_lights.id.g[i] = random_float(0.0f, 1.0f);
		cpu_lights.id.b[i] = random_float(0.0f, 1.0f);
		cpu_lights.pos.x[i] = SPREAD * RATIO * random_float(-1.0f, 1.0f);
		cpu_lights.pos.y[i] = SPREAD * random_float(-1.0f, 1.0f);
		cpu_lights.pos.z[i] = SPREAD * random_float(-1.0f, 1.0f);
	}

	for (int i = 0; i < n; ++i)
	{
		cpu_scene.objects.color.r[i] = random_float(0.0f, 1.0f);
		cpu_scene.objects.color.g[i] = random_float(0.0f, 1.0f);
		cpu_scene.objects.color.b[i] = random_float(0.0f, 1.0f);


		cpu_scene.objects.ka[i] = random_float(0.2f, 0.4f);
		cpu_scene.objects.kd[i] = random_float(0.0f, 0.03f);
		cpu_scene.objects.ks[i] = random_float(0.2f, 0.7f);
		cpu_scene.objects.alpha[i] = random_float(10, 100);
	}

	cpu_scene.objects.primitives.pos.x[0] =
		cpu_scene.objects.primitives.pos.y[0] =
		cpu_scene.objects.primitives.pos.z[0] = 0.0f;


	cpu_scene.objects.primitives.pos.x[1] = 
		cpu_scene.objects.primitives.pos.y[1] =
		cpu_scene.objects.primitives.pos.z[1] = 1.0f;

	cpu_scene.objects.primitives.radius[0] = 1.0f;
	cpu_scene.objects.primitives.radius[1] = 1.0f;

	cpu_scene.csg[0] = 2;
}

void copy_scene_to_gpu(int levels)
{
	int n = pow(2, levels - 1); // spheres count
	int nodes = pow(2, levels) - 1; // nodes count
	int m = LIGHT_COUNT;
	hipEventRecord(start_cpu_gpu, 0);
	// lights copy
	checkCudaErrors(hipMemcpy(gpu_lights.is.r, cpu_lights.is.r, m * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_lights.is.g, cpu_lights.is.g, m * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_lights.is.b, cpu_lights.is.b, m * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_lights.id.r, cpu_lights.id.r, m * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_lights.id.g, cpu_lights.id.g, m * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_lights.id.b, cpu_lights.id.b, m * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_lights.pos.x, cpu_lights.pos.x, m * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_lights.pos.y, cpu_lights.pos.y, m * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_lights.pos.z, cpu_lights.pos.z, m * sizeof(float), hipMemcpyHostToDevice));

	//scene copy
	checkCudaErrors(hipMemcpy(gpu_scene.objects.color.r, cpu_scene.objects.color.r, n * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_scene.objects.color.g, cpu_scene.objects.color.g, n * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_scene.objects.color.b, cpu_scene.objects.color.b, n * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_scene.objects.primitives.pos.x, cpu_scene.objects.primitives.pos.x, n * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_scene.objects.primitives.pos.y, cpu_scene.objects.primitives.pos.y, n * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_scene.objects.primitives.pos.z, cpu_scene.objects.primitives.pos.z, n * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_scene.objects.primitives.radius, cpu_scene.objects.primitives.radius, n * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_scene.objects.ka, cpu_scene.objects.ka, n * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_scene.objects.kd, cpu_scene.objects.kd, n * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_scene.objects.ks, cpu_scene.objects.ks, n * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_scene.objects.alpha, cpu_scene.objects.alpha, n * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_scene.bounding.pos.x, cpu_scene.bounding.pos.x, nodes * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_scene.bounding.pos.y, cpu_scene.bounding.pos.y, nodes * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_scene.bounding.pos.z, cpu_scene.bounding.pos.z, nodes * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_scene.bounding.radius, cpu_scene.bounding.radius, nodes * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_scene.csg, cpu_scene.csg, nodes * sizeof(int), hipMemcpyHostToDevice));
	/*checkCudaErrors(hipMemcpy(&gpu_scene.levels, &cpu_scene.levels, sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(&gpu_scene.count, &cpu_scene.count, sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(&gpu_scene.nodesCount, &cpu_scene.nodesCount, sizeof(int), hipMemcpyHostToDevice));*/

	hipEventRecord(stop_cpu_gpu, 0);
	hipEventSynchronize(stop_render);
	hipEventElapsedTime(&time_cpu_gpu, start_cpu_gpu, stop_cpu_gpu);
}